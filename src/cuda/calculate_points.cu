#include "hip/hip_runtime.h"
#include "calculate_points.h"

__global__ void calc_points_kernel(const float *disp, const float *map_x, const float *map_y, int w, int h, float fx,
                                   float fy, float cx, float cy, float base_line, float *pcd_ptr, bool *valid)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int px = idx % w;
    int py = idx / w;

    if (px >= w || py >= h)
        return;

    float d = disp[py * w + px] / 16.0f;
    int disp_x = __float2int_rd(px - d);

    float lx = map_x[py * w + px];
    float rx = (disp_x >= 0 && disp_x < w) ? map_x[py * w + disp_x] : 0.0f;
    float adjusted_d = lx - rx;

    float cy_val = map_y[py * w + px];

    float z = fx * base_line / adjusted_d;
    float x = z * (lx - cx) / fx;
    float y = z * (cy_val - cy) / fy;

    bool point_valid = (px >= 0) && (py >= 0) && (d >= 1.0f) && (disp_x >= 0) && (disp_x < w) &&
                       (fabsf(adjusted_d) > 1e-6) && isfinite(x) && isfinite(y) && isfinite(z);

    pcd_ptr[3 * (py * w + px) + 0] = point_valid ? x : 0.0f;
    pcd_ptr[3 * (py * w + px) + 1] = point_valid ? y : 0.0f;
    pcd_ptr[3 * (py * w + px) + 2] = point_valid ? z : 0.0f;
    valid[py * w + px] = point_valid;
}

extern "C" void calc_points(const float *disp, const float *map_x, const float *map_y, int w, int h, float fx, float fy,
                            float cx, float cy, float base_line, float *pcd_ptr, bool *valid)
{
    int block_size = 32;
    int grid_size = (w * h + block_size - 1) / block_size;

    calc_points_kernel<<<grid_size, block_size>>>(disp, map_x, map_y, w, h, fx, fy, cx, cy, base_line, pcd_ptr, valid);
    hipDeviceSynchronize();
}
